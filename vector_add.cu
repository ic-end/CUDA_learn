#include "hip/hip_runtime.h"
// 2014 1 14 by lichao
// CUDA实战 27页
// 矢量求和
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <Windows.h>

#define N 10
using namespace std;

__global__ void add( int *a, int *b, int *c)
{
	int tid = blockIdx.x; //计算该索引处的数据
	if ( tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main()
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	hipError_t cudaStatus;
	
	// assign numbers to 'a' and 'b' on the cpu
	for (int i=0; i<N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	// malloc memory on the GPU
	cudaStatus = hipMalloc( (void**)&dev_a, N * sizeof(int) );
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc failed!" << endl;
		goto Error;
	}
	cudaStatus = hipMalloc( (void**)&dev_b, N * sizeof(int) );
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc failed!" << endl;
		goto Error;
	}
	cudaStatus = hipMalloc( (void**)&dev_c, N * sizeof(int) );
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc failed!" << endl;
		goto Error;
	}

	// copy memory from host to device
	cudaStatus = hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy failed!" << endl;
		goto Error;
	}
	cudaStatus = hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy failed!" << endl;
		goto Error;
	}

	add<<<N,1>>>( dev_a, dev_b, dev_c );

	cudaStatus = hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy failed!" << endl;
		goto Error;
	}	
	for (int i=0; i<N; i++)
	{
		cout << "c:" << c[i] << endl;
	}
	
Error:
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	Sleep( 20000 );
	return 0;
}
