#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include "sm_12_atomic_functions.h"

#include <iostream>
#include <Windows.h>
#include <time.h>

#include "cudaErrorYoN.h"

using namespace std;

#define SIZE    (100*1024*1024)
#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

void* big_random_block( int size ) {
    unsigned char *data = (unsigned char*)malloc( size );
    HANDLE_NULL( data );
    for (int i=0; i<size; i++)
        data[i] = rand();

    return data;
}

__global__ void histo_kernel( unsigned char *buffer,
                              long size,
                              unsigned int *histo ) {
    // calculate the starting index and the offset to the next
    // block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd( &histo[buffer[i]], 1 );
        i += stride;
    }
}

int main( void ) {
    unsigned char *buffer =
                     (unsigned char*)big_random_block( SIZE );

    // capture the start time
    // starting the timer here so that we include the cost of
    // all of the operations on the GPU.
    hipEvent_t     start, stop;
    cudaErrorYoN( hipEventCreate( &start ), 4);
    cudaErrorYoN( hipEventCreate( &stop ), 4);
    cudaErrorYoN( hipEventRecord( start, 0 ), 4);

    // allocate memory on the GPU for the file's data
    unsigned char *dev_buffer;
    unsigned int *dev_histo;
    cudaErrorYoN( hipMalloc( (void**)&dev_buffer, SIZE ), 1);
    cudaErrorYoN( hipMemcpy( dev_buffer, buffer, SIZE,
                              hipMemcpyHostToDevice ), 1);

    cudaErrorYoN( hipMalloc( (void**)&dev_histo,
                              256 * sizeof( int ) ), 1);
    cudaErrorYoN( hipMemset( dev_histo, 0,
                              256 * sizeof( int ) ), 1);

    // kernel launch - 2x the number of mps gave best timing
    hipDeviceProp_t  prop;
    cudaErrorYoN( hipGetDeviceProperties( &prop, 0 ), 4);
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2,256>>>( dev_buffer, SIZE, dev_histo );
    
    unsigned int    histo[256];
    cudaErrorYoN( hipMemcpy( histo, dev_histo,
                              256 * sizeof( int ),
                              hipMemcpyDeviceToHost ), 2);

    // get stop time, and display the timing results
    cudaErrorYoN( hipEventRecord( stop, 0 ), 4);
    cudaErrorYoN( hipEventSynchronize( stop ), 4);
    float   elapsedTime;
    cudaErrorYoN( hipEventElapsedTime( &elapsedTime,
                                        start, stop ), 4);
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    long histoCount = 0;
    for (int i=0; i<256; i++) {
        histoCount += histo[i];
    }
    printf( "Histogram Sum:  %ld\n", histoCount );

    // verify that we have the same counts via CPU
    for (int i=0; i<SIZE; i++)
        histo[buffer[i]]--;
    for (int i=0; i<256; i++) {
        if (histo[i] != 0)
            printf( "Failure at %d!  Off by %d\n", i, histo[i] );
    }

    cudaErrorYoN( hipEventDestroy( start ), 4);
    cudaErrorYoN( hipEventDestroy( stop ), 4);
    hipFree( dev_histo );
    hipFree( dev_buffer );
    free( buffer );
	Sleep(2000);
    return 0;
}