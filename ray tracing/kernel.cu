#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>
#include <Windows.h>

#include "cpu_bitmap.h"

#include "cudaErrorYoN.h"

using namespace std;

#define DIM 1024

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

struct Sphere 
{
    float   r,b,g; // �����ɫ
    float   radius;// ��İ뾶
    float   x,y,z; // �����������
	// �ṹ���еĺ��� ������c++�е���
    __device__ float hit( float ox, float oy, float *n ) 
	{
		// �ж�(ox, oy)�����Ƿ��������ཻ
		// ����ཻ������������������������洦�ľ���
		// ��Ϊ����������������ཻʱ��ֻ����ӽ������������ܿ���
        float dx = ox - x;
        float dy = oy - y;
        if (dx*dx + dy*dy < radius*radius) 
		{
            float dz = sqrtf( radius*radius - dx*dx - dy*dy );
            *n = dz / sqrtf( radius * radius );
            return dz + z;
        }
        return -INF;
    }
};

#define SPHERES 30

// __constant__ Sphere s[SPHERES];

__global__ void kernel( Sphere *s, unsigned char *ptr ) 
{
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float   ox = (x - DIM/2);
    float   oy = (y - DIM/2);

    float   r=0, g=0, b=0;
    float   maxz = -INF;
	// �жϹ�����ÿ�������ཻ�����
    for(int i=0; i<SPHERES; i++) 
	{
        float   n;
        float   t = s[i].hit( ox, oy, &n );
        if (t > maxz) 
		{
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    } 

    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock 
{
    unsigned char   *dev_bitmap;
	Sphere          *s;
};



int main( void ) 
{
    DataBlock   data;

    // ��¼��ʼʱ��
    hipEvent_t     start, stop;
    cudaErrorYoN( hipEventCreate( &start ), 4);
    cudaErrorYoN( hipEventCreate( &stop ), 4);
    cudaErrorYoN( hipEventRecord( start, 0 ), 4);

    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char   *dev_bitmap;
	Sphere          *s;

    // ��GPU�Ϸ����ڴ��Լ������λͼ
    cudaErrorYoN( hipMalloc( (void**)&dev_bitmap,
                              bitmap.image_size() ), 1);
    // Ϊ Sphere���ݼ������ڴ�
    cudaErrorYoN( hipMalloc( (void**)&s,
                              sizeof(Sphere) * SPHERES ), 1);

    // ������ʱ�ڴ棬��CPU�϶����ʼ���������Ƶ�GPU�ڴ��ϣ�Ȼ���ͷ��ڴ�
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );

	// ΪSPHERES��Բ����λ�ã���ɫ���뾶��Ϣ
    for (int i=0; i<SPHERES; i++) 
	{
        temp_s[i].r = rnd( 1.0f );
        temp_s[i].g = rnd( 1.0f );
        temp_s[i].b = rnd( 1.0f );
        temp_s[i].x = rnd( 1000.0f ) - 500;
        temp_s[i].y = rnd( 1000.0f ) - 500;
        temp_s[i].z = rnd( 1000.0f ) - 500;
        temp_s[i].radius = rnd( 100.0f ) + 20;
    }

    cudaErrorYoN( hipMemcpy( s, temp_s, 
                                sizeof(Sphere) * SPHERES, 
								hipMemcpyHostToDevice), 2);
    free( temp_s );

    // generate a bitmap from our sphere data
    dim3    grids(DIM/16,DIM/16);
    dim3    threads(16,16);
    kernel<<<grids,threads>>>( s, dev_bitmap );

    // ��λͼ��GPU�ϸ��Ƶ�������
    cudaErrorYoN( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ), 2);

    // ��¼����ʱ��
    cudaErrorYoN( hipEventRecord( stop, 0 ), 4);
    cudaErrorYoN( hipEventSynchronize( stop ), 4);

	// ��ʾ����ʱ��
    float   elapsedTime;
    cudaErrorYoN( hipEventElapsedTime( &elapsedTime, start, stop ), 4); // ���������¼�֮���ʱ��
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

	// �����¼�
    cudaErrorYoN( hipEventDestroy( start ), 4);
    cudaErrorYoN( hipEventDestroy( stop ), 4);

	// �ͷ��ڴ�
    cudaErrorYoN( hipFree( dev_bitmap ), 3);
	cudaErrorYoN( hipFree( s ), 3);

    // ��ʾλͼ
    bitmap.display_and_exit();
}

